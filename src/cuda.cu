#include "hip/hip_runtime.h"

void __global__ cuda_cal1(int * dst, unsigned * src, int n,int num);

void __host__ cuda_caluc1(int * dst, unsigned * src, int n,int num) {
  int *d_dst;
  unsigned *d_src;
  hipMalloc((void**)&d_dst,n*sizeof(int));
  hipMalloc((void**)&d_src,n*sizeof(unsigned));
  hipMemcpy(d_src,src,n*sizeof(unsigned),hipMemcpyHostToDevice);
  cuda_cal1<<<(n/512)+1,512>>>(d_dst,d_src,n,num);
  hipMemcpy(dst,d_dst,n*sizeof(int),hipMemcpyDeviceToHost);
  hipFree(d_dst);
  hipFree(d_src);
}

void __global__ cuda_cal1(int * dst, unsigned * src, int n,int num) {
   unsigned int index = blockIdx.x*blockDim.x+threadIdx.x;
   if(index<n) {
     dst[index] = src[index]+num;
    }
}