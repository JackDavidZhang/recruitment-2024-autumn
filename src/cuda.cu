#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

void __host__ cuda_caluc1(int * dst, unsigned * src, int n,int num) {
  int *d_dst,*d_src;
  hipMalloc((void**)&d_dst,n*sizeof(int));
  hipMalloc((void**)&d_src,n*sizeof(unsigned));
  hipMemcpy(d_src,src,n*sizeof(unsigned),hipMemcpyHostToDevice);
  cuda_cal1<<<n/1024+1,1024>>>(d_dst,d_src,n,num);
  hipMemcpy(dst,d_dst,n*sizeof(int),hipMemcpyDeviceToHost);
}

void __global__ cuda_cal1(int * dst, unsigned * src, int n,int num) {
   int index = blockIdx.x*blockDim.x+threadIdx.x;
   if(index<n) {
     dst[index] = src[index]+num;
    }
}